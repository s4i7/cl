#include "hip/hip_runtime.h"
#include "tensor.cuh"

__global__ auto initCurandStates(hiprandState *states, unsigned long seed, int rows, int cols) -> void {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalElements = rows * cols;
  if (idx < totalElements) {
    hiprand_init(seed, idx, 0, &states[idx]);
  }
}

__global__ auto generateRandomMatrix(float *matrix, hiprandState *states, int rows, int cols) -> void {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalElements = rows * cols;
  if (idx < totalElements) {
    matrix[idx] = hiprand_uniform(&states[idx]);
  }
}

__global__ void check_matrix_equality_atomic(const float* A, const float* B, int n, int m, bool* result, float epsilon) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= n * m) return;

  if (fabsf(A[idx] - B[idx]) > epsilon) {
    atomicExch((int*)result, 0);
  }
}
