#include "device.cuh"
#include "macros.cuh"
#include <cstdio>

auto init_device(bool list_devices) -> int {
  int deviceCount;
  CUDA_CALL(hipGetDeviceCount(&deviceCount));

  if (list_devices) {
    std::printf("Found %d device(s)\n", deviceCount);
    for (int i = 0; i < deviceCount; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      std::printf("Device %d: %s\n", i, prop.name);
    }
  }

  CUDA_CALL(hipSetDevice(0));
  return 0;
}
