#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include "gemm.cuh"
#include "device.cuh"
#include "tensor.cuh"
#include <nvtx3/nvToolsExt.h>
#include <string>
using i64 = int64_t;

constexpr float EPS = 1e-2;

auto get_time() -> i64 {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec * 1000000 + ts.tv_nsec / 1000;
}

auto get_microseconds(hipEvent_t &start, hipEvent_t &stop) -> i64 {
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  return ms * 1000;
}

auto matmul_naive_vs_cublas() -> int {
  nvtxRangePush("matmul");
  int n = 4096;
  int k = 4096;
  int m = 4096;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  nvtxRangePush("matrix memory allocation");
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));
  nvtxRangePop();

  auto run = [&](int iter_num, std::string name, float eps) -> int {
    float alpha = 1.0, beta = 0.0;
    i64 total_smem_time = 0, total_cublas_time = 0, total_tiled_time = 0;

    hiprandState *s_a, *s_b;
    hipMalloc(&s_a, n * k * sizeof(hiprandState));
    hipMalloc(&s_b, k * m * sizeof(hiprandState));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int iter=0; iter<iter_num; iter++) {
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
      initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
      generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

      hipDeviceSynchronize();

      hipEvent_t tiled_start, tiled_end;
      hipEventCreate(&tiled_start);
      hipEventCreate(&tiled_end);
      dim3 gridDim(4096 / 32, 4096 / 32);
      dim3 blockDim(32 * 32);
      nvtxRangePush("gemm_tiled execution");
      hipEventRecord(tiled_start);
      gemm_tiled<32><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);
      hipEventRecord(tiled_end);
      nvtxRangePop();

      hipDeviceSynchronize();

      hipEvent_t smem_start, smem_end;
      hipEventCreate(&smem_start);
      hipEventCreate(&smem_end);
      const int blocksz = 32;
      dim3 gridDimSmem(n / blocksz, m / blocksz);
      dim3 blockDimSmem(blocksz * blocksz);
      nvtxRangePush("gemm_smem execution");
      hipEventRecord(smem_start);
      gemm_tiled_smem<blocksz><<<gridDimSmem, blockDimSmem>>>(d_c, d_a, d_b, n, k, m);
      hipEventRecord(smem_end);
      nvtxRangePop();

      hipDeviceSynchronize();

      hipEvent_t cublas_start, cublas_end;
      hipEventCreate(&cublas_start);
      hipEventCreate(&cublas_end);

      nvtxRangePush("cublas sgemm execution");
      hipEventRecord(cublas_start);
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);
      hipEventRecord(cublas_end);
      nvtxRangePop();

      hipDeviceSynchronize();

      total_smem_time += get_microseconds(smem_start, smem_end);
      total_cublas_time += get_microseconds(cublas_start, cublas_end);
      total_tiled_time += get_microseconds(tiled_start, tiled_end);
    }

    i64 average_smem_time = total_smem_time / iter_num;
    i64 average_cublas_time = total_cublas_time / iter_num;
    i64 average_tiled_time = total_tiled_time / iter_num;
    double average_smem_flops = ((2.0 * n * m * k) / average_smem_time) / 1e3;
    double average_coalesced_flops = ((2.0 * n * m * k) / average_tiled_time) / 1e3;

    std::printf("%s avg smem time: %ld\n", name.c_str(), average_smem_time);
    std::printf("%s avg smem gflops: %lf\n", name.c_str(), average_smem_flops);
    std::printf("%s avg tiled time: %ld\n", name.c_str(), average_tiled_time);
    std::printf("%s avg tiled gflops: %lf\n", name.c_str(), average_coalesced_flops);
    std::printf("%s avg cublas time: %ld\n", name.c_str(), average_cublas_time);

    hipFree(s_a);
    hipFree(s_b);
    hipblasDestroy(handle);
    return 0;
  };

  int warmup_runs = 10;
  int perf_runs = 100;

  if(auto err = run(warmup_runs, "warmup", EPS); err != 0) {
    std::printf("Warmup failed\n");
    return err;
  }
  std::printf("Warmup successfull, naive = cublas\n");

  if (auto err = run(perf_runs, "perf", EPS); err != 0) {
    std::printf("Perf failed\n");
    return err;
  } 
  std::printf("Perf successfull, naive = cublas\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  return 0;
}

auto main() -> int {
  if (auto err = init_device(); err != 0) {
    return err;
  }
  if (auto err = matmul_naive_vs_cublas(); err != 0) {
    return err;
  }
  return 0;
}
