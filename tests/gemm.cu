#include <hipblas.h>
#include <gtest/gtest.h>
#include "device.cuh"
#include "gemm.cuh"
#include "tensor.cuh"

const float eps = 1e-2;

TEST(GEMM_NAIVE_TEST, CheckEqualToCublas) {
  init_device();

  int n = 4096;
  int k = 4096;
  int m = 4096;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));

  float alpha = 1.0, beta = 0.0;

  hiprandState *s_a, *s_b;
  hipMalloc(&s_a, n * k * sizeof(hiprandState));
  hipMalloc(&s_b, k * m * sizeof(hiprandState));

  bool *d_res, h_res;
  hipMalloc(&d_res, sizeof(bool));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

  hipDeviceSynchronize();

  gemm_naive<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_a, d_b, n, k, m);

  hipDeviceSynchronize();

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);

  hipDeviceSynchronize();

  h_res = true;
  hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
  check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_c_ref, n, m, d_res, eps);
  hipDeviceSynchronize();
  hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

  EXPECT_TRUE(h_res);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  hipFree(s_a);
  hipFree(s_b);
  hipFree(d_res);
  hipblasDestroy(handle);
}

TEST(GEMM_TILED_TEST, CheckEqualToCublas) {
  init_device();

  int n = 4096;
  int k = 4096;
  int m = 4096;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));

  float alpha = 1.0, beta = 0.0;

  hiprandState *s_a, *s_b;
  hipMalloc(&s_a, n * k * sizeof(hiprandState));
  hipMalloc(&s_b, k * m * sizeof(hiprandState));

  bool *d_res, h_res;
  hipMalloc(&d_res, sizeof(bool));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

  hipDeviceSynchronize();

  dim3 gridDim(4096 / 32, 4096 / 32);
  dim3 blockDim(32 * 32);

  gemm_tiled<32><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);

  hipDeviceSynchronize();

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);

  hipDeviceSynchronize();

  h_res = true;
  hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
  check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_c_ref, n, m, d_res, eps);
  hipDeviceSynchronize();
  hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

  EXPECT_TRUE(h_res);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  hipFree(s_a);
  hipFree(s_b);
  hipFree(d_res);
  hipblasDestroy(handle);
}

TEST(GEMM_TILED_SMEM_TEST, CheckEqualToCublas) {
  init_device();

  int n = 4096;
  int k = 4096;
  int m = 4096;

  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float *d_a, *d_b, *d_c_ref, *d_c;
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));

  float alpha = 1.0, beta = 0.0;

  hiprandState *s_a, *s_b;
  hipMalloc(&s_a, n * k * sizeof(hiprandState));
  hipMalloc(&s_b, k * m * sizeof(hiprandState));

  bool *d_res, h_res;
  hipMalloc(&d_res, sizeof(bool));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

  hipDeviceSynchronize();

  const int bsz = 32;

  dim3 gridDim(4096 / bsz, 4096 / bsz);
  dim3 blockDim(bsz * bsz);

  gemm_tiled_smem<bsz><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);

  hipDeviceSynchronize();

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);

  hipDeviceSynchronize();

  h_res = true;
  hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
  check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_c_ref, n, m, d_res, eps);
  hipDeviceSynchronize();
  hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

  EXPECT_TRUE(h_res);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  hipFree(s_a);
  hipFree(s_b);
  hipFree(d_res);
  hipblasDestroy(handle);
}
