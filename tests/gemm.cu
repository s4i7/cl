#include <hipblas.h>
#include <gtest/gtest.h>
#include "device.cuh"
#include "gemm.cuh"
#include "tensor.cuh"

const float eps = 1e-2;

template <typename F>
auto vCublas(F &&f) -> bool {
  init_device();

  int n = 4096;
  int k = 4096;
  int m = 4096;

  
  float *d_a, *d_b, *d_c_ref, *d_c;
  hipMalloc(&d_a, n * k * sizeof(float));
  hipMalloc(&d_b, k * m * sizeof(float));
  hipMalloc(&d_c_ref, n * m * sizeof(float));
  hipMalloc(&d_c, n * m * sizeof(float));

  float alpha = 1.0, beta = 0.0;
  int BLOCK_SIZE = 1024;
  int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;

  hiprandState *s_a, *s_b;
  hipMalloc(&s_a, n * k * sizeof(hiprandState));
  hipMalloc(&s_b, k * m * sizeof(hiprandState));

  bool *d_res, h_res;
  hipMalloc(&d_res, sizeof(bool));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_a, time(nullptr), n, k);
  initCurandStates<<<NUM_BLOCKS, BLOCK_SIZE>>>(s_b, time(nullptr), k, m);

  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_a, s_a, n, k);
  generateRandomMatrix<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_b, s_b, k, m);

  hipDeviceSynchronize();

  f(d_a, d_b, d_c, n, k, m);

  hipDeviceSynchronize();

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_b, m, d_a, k, &beta, d_c_ref, m);

  hipDeviceSynchronize();

  h_res = true;
  hipMemcpy(d_res, &h_res, sizeof(bool), hipMemcpyHostToDevice);
  
  check_matrix_equality_atomic<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_c_ref, n, m, d_res, eps);
  hipDeviceSynchronize();
  hipMemcpy(&h_res, d_res, sizeof(bool), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(d_c_ref);
  hipFree(s_a);
  hipFree(s_b);
  hipFree(d_res);
  hipblasDestroy(handle);
  return h_res;
};

TEST(GEMM_NAIVE_TEST, CheckEqualToCublas) {
  auto f = [&](float *d_a, float *d_b, float *d_c, int n, int k, int m) {
    int BLOCK_SIZE = 1024;
    int NUM_BLOCKS = (n * m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    gemm_naive<<<NUM_BLOCKS, BLOCK_SIZE>>>(d_c, d_a, d_b, n, k, m);
  };
  EXPECT_TRUE(vCublas(f));
}

TEST(GEMM_TILED_TEST, CheckEqualToCublas) {
  auto f = [&](float *d_a, float *d_b, float *d_c, int n, int k, int m) -> void {
    dim3 gridDim(4096 / 32, 4096 / 32);
    dim3 blockDim(32 * 32);
    gemm_tiled<32><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);
  };

  EXPECT_TRUE(vCublas(f));
}

TEST(GEMM_TILED_SMEM_TEST, CheckEqualToCublas) {
  auto f = [&](float *d_a, float *d_b, float *d_c, int n, int k, int m) {
    const int bsz = 32;

    dim3 gridDim(4096 / bsz, 4096 / bsz);
    dim3 blockDim(bsz * bsz);

    gemm_tiled_smem<bsz><<<gridDim, blockDim>>>(d_c, d_a, d_b, n, k, m);
  };

  EXPECT_TRUE(vCublas(f));
}

TEST(GEMM_1D_BLOCKTILING_TEST, CheckEqualToCublas) {
  auto f = [&](float *d_a, float *d_b, float *d_c, int n, int k, int m) {
    const int BN = 64;
    const int BK = 8;
    const int BM = 64;
    const int TN = 8;

    dim3 gd(4096 / BN, 4096 / BM);
    dim3 bd((BN * BM) / TN);

    gemm_1d_blocktiling<BN, BK, BM, TN><<<gd, bd>>>(d_c, d_a, d_b, n, k, m);
  };

  EXPECT_TRUE(vCublas(f));
}

TEST(GEMM_2D_BLOCKTILING_TEST, CheckEqualToCublas) {
  auto f = [&](float *d_a, float *d_b, float *d_c, int n, int k, int m) {
    const int BN = 64;
    const int BK = 8;
    const int BM = 64;
    const int TN = 8;
    const int TM = 8;

    dim3 gd(4096 / BN, 4096 / BM);
    dim3 bd((BN / TN) * (BM / TM));

    gemm_2d_blocktiling<BN, BK, BM, TN, TM><<<gd, bd>>>(d_c, d_a, d_b, n, k, m);
  };

  EXPECT_TRUE(vCublas(f));
}
